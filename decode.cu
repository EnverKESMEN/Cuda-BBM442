
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
 
// CUDA kernel. Each thread takes care of one element of c
__global__ void encode(char *encodedText, char *decodedText)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int startEncoded = id * 101;
    int startDecoded = id * 4;
    int t,finish=startEncoded+100;
    // Make sure we do not go out of bounds
    if (id < 15360)
    {
        for(t=startEncoded;t<finish;t++)
        {
            if(encodedText[t]==',')
            {
                decodedText[startDecoded]=encodedText[t+1];
                startDecoded++;
            }
        }
    }
}
 
int main( int argc, char* argv[] )
{
    struct	timeval	stop,	start;
    int decodedSize=15360*4;
    int encodedSize=15360*101;
    // Size of vectors
    int n = 15360;
    int i,j=0;
 
    // Host input vectors
    char *h_encodedText;
    char *h_decodedText;
    char *h_decodedSerialOnCPU;

 
    // Device input vectors
    char *d_encodedText;
    char *d_decodedText;


    // Size, in bytes, of each vector
    size_t bytesOfEncoded = encodedSize*sizeof(char);
    size_t bytesOfDecoded = decodedSize*sizeof(char);
 
    // Allocate memory for each vector on host
    h_encodedText = (char*)malloc(bytesOfEncoded);
    h_decodedText = (char*)malloc(bytesOfDecoded);
    h_decodedSerialOnCPU = (char*)malloc(bytesOfDecoded);
   
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_encodedText, bytesOfEncoded);
    hipMalloc(&d_decodedText, bytesOfDecoded);
    

    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 512;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);


    /* Open your_file in read-only mode */
    FILE *fp = fopen("encodedfile.txt", "r");

    /* Read the file into the buffer */
    fread(h_encodedText, bytesOfEncoded-1, 1, fp); /* Read 1 chunk of size bytes from fp into buffer */

    /* NULL-terminate the buffer */
    h_encodedText[bytesOfEncoded] = '\0';

    gettimeofday(&start,	NULL);
    for(i=0;i<bytesOfEncoded;i++)
    {
    if(h_encodedText[i]==',')
    {
    h_decodedSerialOnCPU[j++]=h_encodedText[i+1];}
    }
    gettimeofday(&stop,	NULL);
    float	SerialElapsed	=	(stop.tv_sec	- start.tv_sec)	*	1000.0f	+	(stop.tv_usec	- start.tv_usec)	/	1000.0f;
    printf("Code	executed	in	%f	milliseconds.\n",	SerialElapsed);

  
  gettimeofday(&start,	NULL);
    // Copy host vectors to device
    hipMemcpy( d_encodedText, h_encodedText, bytesOfEncoded, hipMemcpyHostToDevice);
    //cudaMemcpy( d_decodedText, h_decodedText, bytesOfDecoded, cudaMemcpyHostToDevice);
 
 
    // Execute the kernel
    encode<<<gridSize, blockSize>>>(d_encodedText, d_decodedText);
 
    // Copy array back to host
    hipMemcpy( h_decodedText, d_decodedText, bytesOfDecoded, hipMemcpyDeviceToHost );
    gettimeofday(&stop,	NULL);
    float elapsed	=	(stop.tv_sec	- start.tv_sec)	*	1000.0f	+	(stop.tv_usec	- start.tv_usec)	/	1000.0f;
    printf("Code	executed	in	%f	milliseconds.\n",	elapsed);

    printf("SpeedUp	%f	.\n",	SerialElapsed/elapsed);

    //Write decoded text
    FILE *file = fopen("decodedfile.txt", "w");

    int results = fputs(h_decodedText, file);
    if (results == EOF) {
    // Failed to write do error code here.
    }
    fclose(file);
 
    // Release device memory
    hipFree(d_encodedText);
    hipFree(d_decodedText);
    
 
    // Release host memory
    free(h_encodedText);
    free(h_decodedText);
    
 
    return 0;
}
